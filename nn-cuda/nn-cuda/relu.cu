#include "hip/hip_runtime.h"
#include "common.h"
#define TEST_PYTORTH true

/**
 * @brief
 *  This function performs the forward pass of a ReLU activation function.
 *
 * @param input: Input tensor of shape (B, N) where B is the batch size and N is the number of elements per batch.
 * @param output: Output tensor of the same shape as the input.
 */
__global__ void relu_forward(float *input, float *output, int B, int N)
{
    // This maps one thread to one element in the input.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < B && j < N)
    {
        int idx = i * N + j;
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

/**
 * @brief
 *  This function performs the backward pass of a ReLU activation function.
 *
 * @param input: Input tensor of shape (B, N) from the forward pass.
 * @param grad_output: Gradient tensor from the next layer.
 * @param grad_input: Gradient tensor to propagate back.
 */
__global__ void relu_backward(float *input, float *grad_output, float *grad_input, int B, int N)
{
    // This maps one thread to one element in the input.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < B && j < N)
    {
        int idx = i * N + j;
        grad_input[idx] = input[idx] > 0 ? grad_output[idx] : 0;
    }
}

void relu_forward_cpu(float *input, float *output, int B, int N)
{
    for (int i = 0; i < B; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int idx = i * N + j;
            output[idx] = fmaxf(0.0f, input[idx]);
        }
    }
}

void relu_backward_cpu(float *input, float *grad_output, float *grad_input, int B, int N)
{
    for (int i = 0; i < B; i++)
    {
        for (int j = N - 1; j >= 0; j--)
        {
            int idx = i * N + j;
            grad_input[idx] = input[idx] > 0 ? grad_output[idx] : 0;
        }
    }
}

void runKernel1(float *input, float *output, int B, int N, int sqrt_block_size)
{
    dim3 block(sqrt_block_size, sqrt_block_size);
    dim3 grid((B + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    relu_forward<<<grid, block>>>(input, output, B, N);
    cudaCheck(hipDeviceSynchronize());
}

void runKernel2(float *input, float *grad_output, float *grad_input, int B, int N, int sqrt_block_size)
{
    dim3 block(sqrt_block_size, sqrt_block_size);
    dim3 grid((B + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    relu_backward<<<grid, block>>>(input, grad_output, grad_input, B, N);
    cudaCheck(hipDeviceSynchronize());
}

int main()
{
    srand(0);
    const unsigned long B = 100, N = 100;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // Create host memory of random numbers
    float *out = (float *)malloc(B * N * sizeof(float));
    float *grad_input = (float *)malloc(B * N * sizeof(float));
    float *grad_output = make_random_float(B * N);
    float *inp = make_random_float(B * N);

#if TEST_PYTORTH
    write_npy("relu-layer\\X_relu.npy", inp, 2, new unsigned long[2]{B, N});
    write_npy("relu-layer\\up_grad_relu.npy", grad_output, 2, new unsigned long[2]{B, N});
#endif

    // Move to GPU
    float *d_out, *d_grad_input, *d_grad_output, *d_inp;
    cudaCheck(hipMalloc(&d_out, B * N * sizeof(float)));
    cudaCheck(hipMalloc(&d_grad_input, B * N * sizeof(float)));
    cudaCheck(hipMalloc(&d_grad_output, B * N * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * N * sizeof(float)));

    cudaCheck(hipMemcpy(d_inp, inp, B * N * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_grad_output, grad_output, B * N * sizeof(float), hipMemcpyHostToDevice));

    relu_forward_cpu(inp, out, B, N);
    relu_backward_cpu(inp, grad_output, grad_input, B, N);

#if TEST_PYTORTH
    write_npy("relu-layer\\out_relu.npy", out, 2, new unsigned long[2]{B, N});
    write_npy("relu-layer\\down_grad_relu.npy", grad_input, 2, new unsigned long[2]{B, N});
#endif

    int sqrt_block_sizes[] = {4, 8, 16, 32};

    for (int j = 0; j < sizeof(sqrt_block_sizes) / sizeof(int); j++)
    {
        int sqrt_block_size = sqrt_block_sizes[j];
        printf("Checking block size %d x %d.\n", sqrt_block_size, sqrt_block_size);

        runKernel1(d_inp, d_out, B, N, sqrt_block_size);
        validate_result(d_out, out, "out", B * N, 1e-4f);

        runKernel2(d_inp, d_grad_output, d_grad_input, B, N, sqrt_block_size);
        validate_result(d_grad_input, grad_input, "grad_in", B * N, 1e-4f);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(sqrt_block_sizes) / sizeof(int); j++)
    {
        int sqrt_block_size = sqrt_block_sizes[j];

        int repeat_times = 100;

        float elapsed_time1 = benchmark_kernel(repeat_times, runKernel1, d_inp, d_out, B, N, sqrt_block_size);
        float elapsed_time2 = benchmark_kernel(repeat_times, runKernel2, d_inp, d_grad_output, d_grad_input, B, N, sqrt_block_size);

        float tflops1 = (float)B * N * 1 / elapsed_time1 * 1e3f / 1e12f;
        float tflops2 = (float)B * N * 1 / elapsed_time2 * 1e3f / 1e12f;

        printf("Forward pass: sqrt_block_size %4d | time %.4f ms | tflops %.2f\n", sqrt_block_size, elapsed_time1, tflops1);
        printf("Backward pass: sqrt_block_size %4d | time %.4f ms | tflops %.2f\n", sqrt_block_size, elapsed_time2, tflops2);
    }

    free(out);
    free(inp);
    free(grad_input);
    free(grad_output);

    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_grad_input));
    cudaCheck(hipFree(d_grad_output));

    return 0;
}
