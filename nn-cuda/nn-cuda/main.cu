#include "hip/hip_runtime.h"
#include "common.cuh"
#define TEST_PYTORTH true
#include ""
/**
 * @brief
 *  this performs the forward pass of a linear layer
 * y = x W.T  + b
 *
 * @param X: input tensor of shape (B, N) where B is the batch size and N is the number of input neurons
 * @param W: weight tensor of shape (M, N) where M is the number of output neurons
 * @param bias: bias tensor of shape (M)
 * @param y: output tensor of shape (B, M)
 */
__global__ void linear_layer_forward_naive(float* X, float* W, float* bias, float* y, int B, int N, int M)
{
	// this maps one thread to one output element
	// the grid size is (B,M,1)
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// will be used to store the dot product of the i-th row of X and the j-th row of W
	if (i < B && j < M)
	{
		float dot_product = bias[j];
		for (unsigned long k = 0; k < N; k++)
		{
			dot_product += X[i * N + k] * W[j * N + k];
		}
		// store the result in y with the bias
		y[i * M + j] = dot_product;
	}
}

void linear_layer_forward_cpu(float* X, float* W, float* bias, float* y, int B, int N, int M)
{

	for (int i = 0; i < B; i++)
	{
		for (int j = 0; j < M; j++)
		{
			y[i * M + j] = bias[j];
			for (int k = 0; k < N; k++)
			{
				y[i * M + j] += X[i * N + k] * W[j * N + k];
			}
		}
	}
}
void linear_layer(float* X, float* W, float* bias, float* y, int B, int N, int M, int sqrt_block_size)
{
	dim3 block(sqrt_block_size, sqrt_block_size);
	dim3 grid((B + block.x - 1) / block.x, (M + block.y - 1) / block.y);
	linear_layer_forward_naive << <grid, block >> > (X, W, bias, y, B, N, M);
	cudaCheck(hipDeviceSynchronize());
}
/**
 * @brief
 *  This function performs the forward pass of a ReLU activation function.
 *
 * @param input: Input tensor of shape (B, N) where B is the batch size and N is the number of elements per batch.
 * @param output: Output tensor of the same shape as the input.
 */
__global__ void relu_forward(float* input, float* output, int B, int N)
{
	// This maps one thread to one element in the input.
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < B && j < N)
	{
		int idx = i * N + j;
		output[idx] = fmaxf(0.0f, input[idx]);
	}
}
void relu_forward_cpu(float* input, float* output, int B, int N)
{
	for (int i = 0; i < B; i++)
	{
		for (int j = 0; j < N; j++)
		{
			int idx = i * N + j;
			output[idx] = fmaxf(0.0f, input[idx]);
		}
	}
}
void run_relu_kernel(float* input, float* output, int B, int N, int sqrt_block_size)
{
	dim3 block(sqrt_block_size, sqrt_block_size);
	dim3 grid((B + block.x - 1) / block.x, (N + block.y - 1) / block.y);
	relu_forward << <grid, block >> > (input, output, B, N);
	cudaCheck(hipDeviceSynchronize());
}

//-----------------------------------------------------------------------------------
// ------------------------------- cpu version -------------------------------
/**
 * @brief
 *  this a template function to be suitable for float and double numbers to apply softmax
 *
 * @param in: input tensor of shape (N, C) where N is the batch size (number of rows) and C (number of columns) is the number of classes
 * @param out: output tensor of shape (N, C) where N is the batch size (number of rows) and C (number of columns) is the number of classes
 * @param N: number of rows
 * @param C: number of columns
 */
template <class T>
void softmax_cpu(const T* in, T* out, int N, int C)
{
	// loop over each row. each row will get softmaxed
	for (int i = 0; i < N; i++)
	{
		// assume that the first element in the row is the maximum
		T max_val = in[i * C];
		// loop to get the maximum value of each row
		for (int j = 1; j < C; j++)
		{
			if (in[i * C + j] > max_val)
			{
				max_val = in[i * C + j];
			}
		}

		T sum = 0;
		// loop over the row to calculate the sum and apply normalization
		for (int j = 0; j < C; j++)
		{
			// apply normalization step to ensure that the maximum value will be 0 to avoid overflow
			out[i * C + j] = exp(in[i * C + j] - max_val);
			sum += out[i * C + j];
		}
		// output softmaxed values
		for (int j = 0; j < C; j++)
		{
			out[i * C + j] /= sum;
		}
	}
}

// ------------------------------- gpu version -------------------------------
/* each thread will process only one row */
template <class T>
__global__ void softmax_kernel(const T* in_h, T* out_d, int N, int C)
{
	// input dimension (N,C)
	// output dimension (N,C)
	// get actual index in in_h and out_d
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		T max_val = in_h[i * C];
		for (int j = 1; j < C; j++)
		{
			if (in_h[i * C + j] > max_val)
			{
				max_val = in_h[i * C + j];
			}
		}

		T sum = 0;
		for (int j = 0; j < C; j++)
		{
			// apply normalization step to ensure that the maximum value will be 0 to avoid overflow
			out_d[i * C + j] = exp(in_h[i * C + j] - max_val);
			sum += out_d[i * C + j];
		}
		// output softmaxed values
		for (int j = 0; j < C; j++)
		{
			out_d[i * C + j] /= sum;
		}
	}
}

template <class T>
void run_softmax_kernel(const T* input, T* output, int N, int C, int block_size)
{
	int num_blocks = ceil_div(N, block_size);
	softmax_kernel << <num_blocks, block_size >> > (input, output, N, C);
}
//-----------------------------------------------------------------------------------
/**
 * @brief
 *  this is a template function to perform NLL loss
 *  its input is the probabilities from the softmax and the targets
 *
 * @param losses: output tensor of shape (N)
 * @param input: input tensor of shape (N, C) where N is the batch size (number of rows) and C (number of columns) is the number of classes
 * @param targets: target tensor of shape (N) contains number from 0 to C-1
 * @param N: number of rows
 * @param C: number of columns
 */
template <class T>
void cross_entropy_cpu(T* losses, const T* input, const int* targets, int N, int C)
{
	// output: losses is (N) of the individual losses for each batch
	// input: input are (N,C) of the probabilities from softmax
	// input: targets is (N) of integers giving the correct index in logits
	for (int i = 0; i < N; i++)
	{
		losses[i] = -log(input[i * C + targets[i]]);
	}
}

// kernel for cross_entropy
template <class T>
__global__ void cross_entropy_kernel(T* losses, const T* input, const int* targets, int N, int C)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
	{
		losses[i] = -log(input[i * C + targets[i]]);
	}
}

template <class T>
void run_cross_entropy_kernel(T* losses, const T* probs, const int* targets, int N, int C, const int block_size)
{
	const int grid_size = ceil_div(N, block_size);
	cross_entropy_kernel << <grid_size, block_size >> > (losses, probs, targets, N, C);
	cudaCheck(hipGetLastError());
}
//-----------------------------------------------------------------------------------
template <class T>
__global__ void array_sum_kernel3(T* d_a, T* d_result, int size)
{
	extern __shared__ T v[];
	int amount_per_thread = (size + blockDim.x - 1) / blockDim.x;
	int start_index = threadIdx.x * amount_per_thread;
	int end_index = min(start_index + amount_per_thread, size);
	T partialsum = 0.0f;
	for (int k = start_index; k < end_index; k++)
	{
		partialsum += d_a[k];
		v[threadIdx.x] = partialsum;
	}
	__syncthreads();

	/*
	The loop starts with `s` equal to half the block size (`blockDim.x`).
	In each iteration of the loop, each thread with an index less than `s` adds the element at position `threadIdx.x + s` to the element at position `threadIdx.x` in the array `v`.
	The operation `s>>=1` halves `s` in each iteration, effectively reducing the active size of the array by half in each step.
	After each step, `__syncthreads()` is called to ensure that all threads have completed their computations before the next iteration begins. This is necessary because in the next iteration, some threads will be working with results computed by other threads in the current iteration.
	This process continues until `s` becomes 0, at which point all elements of the array have been added together and the total is stored in `v[0]`.
	*/
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (threadIdx.x < s)
		{
			v[threadIdx.x] += v[threadIdx.x + s];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0)
	{
		d_result[0] = v[0];
	}
}

template <class T>
void run_array_sum_kernel3(T* d_a, T* d_result, int size, int block_size)
{
	int num_blocks = ceil_div(size, block_size);
	array_sum_kernel3 << <1, num_blocks, block_size * sizeof(T) >> > (d_a, d_result, size);
	cudaCheck(hipGetLastError());
}

/*
 * @brief
 * This will include the model parameters like weights and bias for each layer
 */
#define NUM_PARAMETER_ARRAYS 4
#define NUM_ACTIVATION_ARRAYS 6
typedef struct
{
	float* ln1w; // linear layer 1 weights (H1 x N)
	float* ln1b; // linear layer 1 bias (H1)
	float* ln2w; // linear layer 2 weights (H2 x H1)
	float* ln2b; // linear layer 2 bias (H2)
} ModelParameters;

/*
 * @brief
 * This will include the model activations like the output of each layer
 */
typedef struct
{
	float* ln1;          // linear layer 1 output (B x H1)
	float* a1;           // activation 1 output (B x H1)
	float* ln2;          // linear layer 2 output (B x H2) -- H2 is the number of classes = C
	float* sm;           // softmax output (B x C)
	float* loss;         // loss (B)
	float* reduced_loss; // reduced loss (1)
} ModelActivation;

typedef struct
{
	unsigned long param_sizes[NUM_PARAMETER_ARRAYS];
	ModelParameters* params;
	float* params_memory;

	unsigned long activation_sizes[NUM_ACTIVATION_ARRAYS];
	ModelActivation* activations;
	float* activations_memory;
} TwoLayerModel;

typedef enum
{
	PARAMETERS_TYPE,
	ACTIVATIONS_TYPE
} DataType;

typedef enum
{
	ZEROS_V,
	ONES_V,
	RANDOM_V
} INITIAL_VALUE_TYPE;

float* float_cpu_malloc_and_point(void* data, unsigned long* sizes, int num_arrays, DataType type, INITIAL_VALUE_TYPE initial_value = ZEROS_V)
{
	unsigned long total_size = 0;
	for (int i = 0; i < num_arrays; i++)
	{
		total_size += sizes[i];
	}
	float* memory;// = (float*)malloc(total_size * sizeof(float));

	switch (initial_value)
	{
	case ZEROS_V:
		memory = make_zeros_float(total_size);
		break;
	case ONES_V:
		memory = make_ones_float(total_size);
		break;
	case RANDOM_V:
		memory = make_random_float(total_size);
		break;
	}
	if (memory == nullptr)
	{
		// Handle allocation failure
		exit(EXIT_FAILURE);
		//return NULL;
	}

	ModelParameters* params = (ModelParameters*)data;
	float** ptrs[] = { &(params->ln1w),
					  &(params->ln1b),
					  &(params->ln2w),
					  &(params->ln2b) };
	float* memory_iterator = memory;
	for (int i = 0; i < num_arrays; i++)
	{
		*(ptrs[i]) = memory_iterator;
		memory_iterator += sizes[i];
	}

	//case ACTIVATIONS_TYPE: // ModelActivation
	//{
	//	ModelActivation* activations = (ModelActivation*)data;
	//	float** ptrs[] = { &activations->ln1,
	//					  &activations->a1,
	//					  &activations->ln2,
	//					  &activations->sm,
	//					  &activations->loss,
	//					  &activations->reduced_loss };
	//	float* memory_iterator = memory;
	//	for (int i = 0; i < num_arrays; i++)
	//	{
	//		*(ptrs[i]) = memory_iterator;
	//		memory_iterator += sizes[i];
	//	}
	//}
	//break;

	return memory;
}

int main()
{
	unsigned long input_dim = 3;
	unsigned long B = 2;
	unsigned long H1 = 3;
	unsigned long H2 = 3;
	unsigned long C = 3;
	TwoLayerModel model;

	model.param_sizes[0] = H1 * input_dim; // ln1w
	model.param_sizes[1] = H1;             // ln1b
	model.param_sizes[2] = H2 * H1;        // ln2w
	model.param_sizes[3] = H2;             // ln2b
	model.params_memory = float_cpu_malloc_and_point(&(model.params), model.param_sizes, NUM_PARAMETER_ARRAYS, PARAMETERS_TYPE);

	if (model.params_memory == NULL)
	{
		// Handle allocation failure
		printf("Allocation failure\n");
		return 1;
	}
	// Now Activations
	model.activation_sizes[0] = B * H1; // ln1
	model.activation_sizes[1] = B * H1; // a1
	model.activation_sizes[2] = B * H2; // ln2
	model.activation_sizes[3] = B * C;  // sm
	model.activation_sizes[4] = B;      // loss
	model.activation_sizes[5] = 1;      // reduced_loss
	model.activations_memory = float_cpu_malloc_and_point(&(model.activations), model.activation_sizes, NUM_ACTIVATION_ARRAYS, ACTIVATIONS_TYPE);

	int deviceIdx = 0;
	cudaCheck(hipSetDevice(deviceIdx));

	// create host memory of random numbers

	float* inp = make_random_float(B * input_dim);
	int* target = make_random_int(B, int(C));

	// #if TEST_PYTORTH
	//     write_npy("all-model\\X_c.npy", inp, 2, new unsigned long[2]{B, input_dim});
	//     write_npy("all-model\\target.npy", target, 1, new unsigned long[1]{B});
	//     write_npy("all-model\\ln1w.npy", (model.params)->ln1w, 2, new unsigned long[2]{H1, input_dim});
	//     write_npy("all-model\\ln1b.npy", model.params->ln1b, 1, new unsigned long[1]{H1});
	//     write_npy("all-model\\ln2w.npy", model.params->ln2w, 2, new unsigned long[2]{H2, H1});
	//     write_npy("all-model\\ln2b.npy", model.params->ln2b, 1, new unsigned long[1]{H2});
	// #endif
}