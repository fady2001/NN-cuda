#include "ModelMemoryHandler.cuh"
#include "common.hpp"
#include "kernels_launchers.cuh"
#include <cmath>
#define TEST_PYTORTH true

int main() {
  uint input_dim = 200;
  uint B = 320;
  uint H1 = 2150;
  uint C = 14;
  ModelMemoryHandler h_model(input_dim, B, H1, C, RANDOM_V, RANDOM_V);

  // create host memory of random numbers
  float *inp = make_random_float(B * input_dim);
  int *target = make_random_int(B, int(C));

#if TEST_PYTORTH
  write_npy("all-model\\X_c.npy", inp, 2, new unsigned long[2]{B, input_dim});
  write_npy("all-model\\target.npy", target, 1, new unsigned long[1]{B});
  write_npy("all-model\\ln1w.npy", h_model.GetParams().ln1w, 2,
            new unsigned long[2]{H1, input_dim});
  write_npy("all-model\\ln1b.npy", h_model.GetParams().ln1b, 1,
            new unsigned long[1]{H1});
  write_npy("all-model\\ln2w.npy", h_model.GetParams().ln2w, 2,
            new unsigned long[2]{C, H1});
  write_npy("all-model\\ln2b.npy", h_model.GetParams().ln2b, 1,
            new unsigned long[1]{C});
#endif

  int deviceIdx = 0;
  cudaCheck(hipSetDevice(deviceIdx));

  // move to GPU
  ModelMemoryHandler d_model;
  h_model.model_to_cuda(&d_model);

  // move input and target to GPU
  float *d_inp;
  uint *d_target;
  cudaCheck(hipMalloc(&d_inp, B * input_dim * sizeof(float)));
  cudaCheck(hipMalloc(&d_target, B * sizeof(uint)));
  cudaCheck(hipMemcpy(d_inp, inp, B * input_dim * sizeof(float),
                       hipMemcpyHostToDevice));
  cudaCheck(
      hipMemcpy(d_target, target, B * sizeof(int), hipMemcpyHostToDevice));

  // run the model
  KernelsLaunchers::linear_layer(
      d_inp, d_model.GetParams().ln1w, d_model.GetParams().ln1b,
      d_model.GetActivations().ln1, B, input_dim, H1, 32);
  save_2d("all-model\\ln1.npy", d_model.GetActivations().ln1, B, H1);
  KernelsLaunchers::run_relu_kernel(d_model.GetActivations().ln1,
                                    d_model.GetActivations().a1, B, H1, 32);
  save_2d("all-model\\a1.npy", d_model.GetActivations().a1, B, H1);
  KernelsLaunchers::linear_layer(
      d_model.GetActivations().a1, d_model.GetParams().ln2w,
      d_model.GetParams().ln2b, d_model.GetActivations().ln2, B, H1, C, 32);
  save_2d("all-model\\ln2.npy", d_model.GetActivations().ln2, B, C);
  KernelsLaunchers::run_softmax_kernel(d_model.GetActivations().ln2,
                                       d_model.GetActivations().sm, B, C, 32);
  save_2d("all-model\\sm.npy", d_model.GetActivations().sm, B, C);
  KernelsLaunchers::run_cross_entropy_kernel(d_model.GetActivations().loss,
                                             d_model.GetActivations().sm,
                                             d_target, B, C, 32);
  save_1d("all-model\\loss.npy", d_model.GetActivations().loss, B);
  KernelsLaunchers::run_reduce_kernel3(d_model.GetActivations().loss,
                                       d_model.GetActivations().reduced_loss, B,
                                       REDUCTION::MEAN, 32);

  // cuda synchronize();
  cudaCheck(hipDeviceSynchronize());
  // copy the loss to the host
  float *reduced_loss = (float *)malloc(sizeof(float));
  cudaCheck(hipMemcpy(reduced_loss, d_model.GetActivations().reduced_loss,
                       sizeof(float), hipMemcpyDeviceToHost));
  printf("Loss: %f\n", *reduced_loss);

  // backpropagation
  KernelsLaunchers::run_crossentropy_softmax_backward(
      d_model.GetDownstreamGradients().dsm, d_model.GetActivations().sm,
      d_target, B, C, 32);
  save_2d("all-model\\dsm.npy", d_model.GetDownstreamGradients().dsm, B, C);
  // get_from_gpu_and_print("dsm", d_model.GetDownstreamGradients().dsm, B * C);

  KernelsLaunchers::runLinearBackward(
      d_model.GetActivations().a1, d_model.GetParams().ln2w,
      d_model.GetDownstreamGradients().dsm, d_model.GetGradients().ln2w_grad,
      d_model.GetGradients().ln2b_grad, d_model.GetDownstreamGradients().dln2,
      B, H1, C, 32);
  save_2d("all-model\\dln2.npy", d_model.GetDownstreamGradients().dln2, B, H1);
  save_2d("all-model\\ln2w_grad.npy", d_model.GetGradients().ln2w_grad, C, H1);
  save_1d("all-model\\ln2b_grad.npy", d_model.GetGradients().ln2b_grad, C);

  // get_from_gpu_and_print("dln2", d_model.GetDownstreamGradients().dln2, B *
  // H1); get_from_gpu_and_print("ln2w_grad", d_model.GetGradients().ln2w_grad,
  // C * H1); get_from_gpu_and_print("ln2b_grad",
  // d_model.GetGradients().ln2b_grad, C);

  KernelsLaunchers::runReluBackward(
      d_model.GetActivations().ln1, d_model.GetDownstreamGradients().dln2,
      d_model.GetDownstreamGradients().da1, B, H1, 32);
  save_2d("all-model\\da1.npy", d_model.GetDownstreamGradients().da1, B, H1);
  // get_from_gpu_and_print("da1", d_model.GetDownstreamGradients().da1, B *
  // H1);
  KernelsLaunchers::runLinearBackward(
      d_inp, d_model.GetParams().ln1w, d_model.GetDownstreamGradients().da1,
      d_model.GetGradients().ln1w_grad, d_model.GetGradients().ln1b_grad,
      d_model.GetDownstreamGradients().dln1, B, input_dim, H1, 32);
  save_2d("all-model\\dln1.npy", d_model.GetDownstreamGradients().dln1, B,
          input_dim);
  save_2d("all-model\\ln1w_grad.npy", d_model.GetGradients().ln1w_grad, H1,
          input_dim);
  save_1d("all-model\\ln1b_grad.npy", d_model.GetGradients().ln1b_grad, H1);
  // get_from_gpu_and_print("dln1", d_model.GetDownstreamGradients().dln1,B *
  // input_dim);

  // Magic optimizer
  KernelsLaunchers::SGD_run_kernel(d_model.GetParamsMemory(),
                                   d_model.GetGradientsMemory(),
                                   d_model.get_num_parameters(), 0.01, 0.0, 32);

#if TEST_PYTORTH
  save_2d("all-model\\updated_ln1w.npy", d_model.GetParams().ln1w, H1,
          input_dim);
  save_1d("all-model\\updated_ln1b.npy", d_model.GetParams().ln1b, H1);
  save_2d("all-model\\updated_ln2w.npy", d_model.GetParams().ln2w, C, H1);
  save_1d("all-model\\updated_ln2b.npy", d_model.GetParams().ln2b, C);
#endif
  return 0;
}
