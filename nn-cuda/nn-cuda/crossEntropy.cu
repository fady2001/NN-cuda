#include "hip/hip_runtime.h"
#include "common.cuh"
#include ""
#include <cmath>
#include <iostream>
#define TEST_PYTORTH true
/**
* @brief
*  this is a template function to perform NLL loss
*  its input is the probabilities from the softmax and the targets
*
* @param losses: output tensor of shape (N)
* @param input: input tensor of shape (N, C) where N is the batch size (number of rows) and C (number of columns) is the number of classes
* @param targets: target tensor of shape (N) contains number from 0 to C-1
* @param N: number of rows
* @param C: number of columns
*/
template<class T>
void cross_entropy_cpu(T* losses,const T* input, const int* targets,int N, int C) {
	// output: losses is (N) of the individual losses for each batch
	// input: input are (N,C) of the probabilities from softmax
	// input: targets is (N) of integers giving the correct index in logits
	for (int i = 0; i < N; i++) {
		losses[i] = -log(input[i * C + targets[i]]);
	}
}

// kernel for cross_entropy
template<class T>
__global__ void cross_entropy_kernel(T* losses, const T* input, const int* targets, int N, int C) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		losses[i] = -log(input[i * C + targets[i]]);
	}
}

template <class T>
void run_cross_entropy_kernel(T *losses, const T *probs, const int *targets, int N, int C, const int block_size)
{
   const int grid_size = ceil_div(N, block_size);
   cross_entropy_kernel<<<grid_size, block_size>>>(losses, probs, targets, N, C);
   cudaCheck(hipGetLastError());
}

int main()
{
	srand(0);
	float* h_losses;
	float* h_predictions;
	int* h_targets;
	const unsigned long C = 100;
	const unsigned long N = 100;
	
	int deviceIdx = 0;
	cudaCheck(hipSetDevice(deviceIdx));

	h_losses = (float*)malloc(N * sizeof(float));
	h_predictions = (float*)malloc(N * C * sizeof(float));
	h_targets = (int*)malloc(N * sizeof(int));

	h_targets = make_random_int(N, C);
	h_predictions = make_random_float_01(N * C);

	// make the input less uniformly random: Otherwise, all probabilities will be basically zero,
	// and the tests are not actually meaningful.
	const int* outliers = make_random_int(N * 3, C);
	for (int k = 0; k < 3; ++k) {
		for (int j = 0; j < N; ++j) {
			h_predictions[j * C + outliers[j * 3 + k]] *= 20;
		}
	}

#if TEST_PYTORTH
   write_npy("cross-entropy-layer\\h_predictions.npy", h_predictions, 2, new size_t[2]{N, C});
   write_npy("cross-entropy-layer\\h_targets.npy", h_targets, 1, new size_t[1]{N});
#endif

	// move to GPU
	float* d_losses;
	float* d_predictions;
	int* d_targets;
	cudaCheck(hipMalloc(&d_losses, N * sizeof(float)));
	cudaCheck(hipMalloc(&d_predictions, N * C * sizeof(float)));
	cudaCheck(hipMalloc(&d_targets, N * sizeof(int)));
	cudaCheck(hipMemcpy(d_predictions, h_predictions, N * C * sizeof(float), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_targets, h_targets, N * sizeof(int), hipMemcpyHostToDevice));

	cross_entropy_cpu<float>(h_losses, h_predictions, h_targets, N, C);
#if TEST_PYTORTH
  write_npy("cross-entropy-layer\\h_losses.npy", h_losses, 1, new size_t[1]{N});
#endif

	// run the kernel
	int block_sizes[] = { 32, 64, 128, 256, 512, 1024 };
	// first check the correctness of the kernel
	for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
		int block_size = block_sizes[j];
		printf("Checking block size %d.\n", block_size);
		crossentropy_forward1(d_losses, d_predictions, d_targets, N, C, block_sizes[j]);
		validate_result(d_losses, h_losses, "out", N, 1e-4f);
	}

	printf("All results match. Starting benchmarks.\n\n");
	for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
		int block_size = block_sizes[j];

		int repeat_times = 100;
		float elapsed_time = benchmark_kernel(repeat_times, crossentropy_forward1<float>, d_losses, d_predictions, d_targets, N, C, block_sizes[j]);

		printf("block_size %4d | time %.4f ms | per token %.2f �s\n", block_size, elapsed_time, elapsed_time * 1'000 / (N * C));
	}

	//free memory
	free(h_losses);
	free(h_predictions);
	free(h_targets);
	cudaCheck(hipFree(d_losses));
	cudaCheck(hipFree(d_predictions));
	cudaCheck(hipFree(d_targets));
	return 0;
}