#include "hip/hip_runtime.h"
#include "common.cuh"
#define TEST_PYTORTH true

/**
 * @brief
 *  this performs the forward pass of a linear layer
 * y = x W.T  + b
 *
 * @param X: input tensor of shape (B, N) where B is the batch size and N is the number of input neurons
 * @param W: weight tensor of shape (M, N) where M is the number of output neurons
 * @param bias: bias tensor of shape (M)
 * @param y: output tensor of shape (B, M)
 */
__global__ void linear_layer_forward_naive(float *X, float *W, float *bias, float *y, int B, int N, int M)
{
    // this maps one thread to one output element
    // the grid size is (B,M,1)
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // will be used to store the dot product of the i-th row of X and the j-th row of W
    if (i < B && j < M)
    {
        float dot_product = bias[j];
        for (size_t k = 0; k < N; k++)
        {
            dot_product += X[i * N + k] * W[j * N + k];
        }
        // store the result in y with the bias
        y[i * M + j] = dot_product;
    }
}

void linear_layer_forward_cpu(float *X, float *W, float *bias, float *y, int B, int N, int M)
{

    for (int i = 0; i < B; i++)
    {
        for (int j = 0; j < M; j++)
        {
            y[i * M + j] = bias[j];
            for (int k = 0; k < N; k++)
            {
                y[i * M + j] += X[i * N + k] * W[j * N + k];
            }
        }
    }
}
void runKernel1(float *X, float *W, float *bias, float *y, int B, int N, int M, int sqrt_block_size)
{
    dim3 block(sqrt_block_size, sqrt_block_size);
    dim3 grid((B + block.x - 1) / block.x, (M + block.y - 1) / block.y);
    linear_layer_forward_naive<<<grid, block>>>(X, W, bias, y, B, N, M);
    cudaCheck(hipDeviceSynchronize());
}
int main()
{
    srand(0);
    const unsigned long B = 100, N = 100, M = 30;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // create host memory of random numbers
    float *out = (float *)malloc(B * M * sizeof(float));
    float *inp = make_random_float(B * N);
    float *weight = make_random_float(M * N);
    float *bias = make_random_float(M);

// write arrays to npy files if you want to test with torch
#if TEST_PYTORTH
    write_npy("X_c.npy", inp, 2, new unsigned long[2]{B, N});
    write_npy("W_C.npy", weight, 2, new unsigned long[2]{M, N});
    write_npy("bias_C.npy", bias, 1, new unsigned long[1]{M});
#endif

    // move to GPU
    float *d_out;
    float *d_inp;
    float *d_weight;
    float *d_bias;
    cudaCheck(hipMalloc(&d_out, B * M * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * N * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, M * N * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, M * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * N * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, M * N * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, bias, M * sizeof(float), hipMemcpyHostToDevice));

    linear_layer_forward_cpu(inp, weight, bias, out, B, N, M);

#if TEST_PYTORTH
    write_npy("out_C.npy", out, 2, new unsigned long[2]{B, M});
#endif

    // print_2D_Matrix(out, "out", B, M);
    int sqrt_block_sizes[] = {4, 8, 16, 32};
    // first check the correctness of the kernel
    for (int j = 0; j < sizeof(sqrt_block_sizes) / sizeof(int); j++)
    {
        int sqrt_block_size = sqrt_block_sizes[j];
        printf("Checking block size %d x %d.\n", sqrt_block_size, sqrt_block_size);
        runKernel1(d_inp, d_weight, d_bias, d_out, B, N, M, sqrt_block_size);
        validate_result(d_out, out, "out", B * M, 1e-4f);
    }

    printf("All results match. Starting benchmarks.\n\n");
    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(sqrt_block_sizes) / sizeof(int); j++)
    {
        int sqrt_block_size = sqrt_block_sizes[j];

        int repeat_times = 100;
        float elapsed_time = benchmark_kernel(repeat_times, runKernel1, d_inp, d_weight, d_bias, d_out, B, N, M, sqrt_block_size);

        // napkin math: estimate the flops achieved
        // e.g. A100 40GB PCIe is advertised at 19.5 TFLOPS fp32
        float tflops = (float)B * N * M * 2 / elapsed_time * 1e3f / 1e12f;
        printf("sqrt_block_size %4d | time %.4f ms | tflops %.2f\n", sqrt_block_size, elapsed_time, tflops);
    }

    // free memory
    free(out);
    free(inp);
    free(weight);
    free(bias);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));
    return 0;
}